/*
 *
 *  Last name: Bora
 *  First name: Anuj
 *  Net ID: aab688
 *
 */


/*
 * This file contains the code for finding prime numbers till N using GPU
 * It uses device 1 in the cluster
 * You compile with:
 * 	  nvcc -o genprimes genprimes.cu --generate-code arch=compute_30,code=sm_30
 *
 * And run as :
 *    time ./genprimes 100
 */

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime_api.h>

#define TILE 64

/*****************************************************************/

// Function declarations:
void  gpu_genprimes(char*, unsigned int);

/*****************************************************************/

/*********************  Kernel  *********************************/

__global__
void warmUpGPU()
{
  // do nothing
}

__global__
void checkEven(char* primes, unsigned int N)
{
   int index = blockIdx.x * blockDim.x + threadIdx.x;
   if (index > 2 && index <= N && (index % 2 == 0))
   {
     primes[index] = '0';
   }
}

__global__
void checkOdd(char* primes, unsigned int N, unsigned int ceiling)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if ((index <= ceiling) && (primes[index] == '1'))
  {
    for (int i = index * 2; i <= N; i = i + index)
    {
       primes[i] = '0';
    }
  }

}

/*****************************************************************/

int main(int argc, char * argv[])
{
  //cudaProfilerStart();
  unsigned int N;

  /* The 1D array of chars will be treated as 1D array of N elements */
  char* numbers;

  if(argc != 2)
  {
    fprintf(stderr, "N: number till which the primes should be generated\n");
    exit(1);
  }
  char* filename = argv[1];
  N = (unsigned int) atoi(argv[1]);

  if (N < 2)
  {
    printf("Zero prime numbers till %d", N);
    printf("\nNo file created");
    return 0;
  }

  numbers = (char*)malloc((N + 1) * sizeof(char));

  if( !numbers )
  {
   fprintf(stderr, " Cannot allocate the %u array\n", N);
   exit(1);
  }

  for (int i = 0; i <= N; i++)
  {
    numbers[i] = '1';
  }

  /**
   *  Initialize 0 and 1 as non-prime numbers
   */
  numbers[0] = '0';
  numbers[1] = '0';

  printf("Generating Prime Numbers ...\n");
  gpu_genprimes(numbers, N);

  FILE *fp;
  const char* extension = ".txt";
  char* file = (char *) malloc(1 + strlen(filename)+ strlen(extension) );
  strcpy(file, filename);
  strcat(file, extension);
  fp = fopen(file, "w+");

  int count = 0;
  for (int i = 2; i <= N; i++)
  {
    if (numbers[i] == '1')
    {
      //fprintf(fp, "%d ", i);
      //printf("%d", i);
      count++;
    }
  }
  fclose(fp);
  printf("Number of Prime Numbers = %d \n", count);
  free(numbers);
  //cudaProfilerStop();
  return 0;
}

/***************** The GPU version  *********************/
void  gpu_genprimes(char* numbers, unsigned int N)
{
  hipSetDevice(1);
  /**
   *  First kernel takes more time to initialize.
   *  Send a kernel which does nothing which will warm up
   *  the GPU and in that time do other work on CPU.
   */
  warmUpGPU<<<1, 1>>>();


  int size = (N + 1) * sizeof(char);
  char* d_numbers;
  unsigned int ceiling = ceil((N + 1) / 2);

  // Made use of 1d blocks and threads as it will make it easier to work on data
  // represented as 1d
  dim3 numBlocks(ceil(1.0*(N + 1)/TILE), 1, 1);
  dim3 threadsPerBlock(TILE, 1, 1);


  // Step 1 : Allocate Memory on Device and copy values to device
  hipMalloc((void **) &d_numbers, size);

  hipMemcpy(d_numbers, numbers, size, hipMemcpyHostToDevice);

  // Step 2 : Launch Kernels
  checkEven<<<numBlocks, threadsPerBlock>>>(d_numbers, N);
  checkOdd<<<numBlocks, threadsPerBlock>>>(d_numbers, N, ceiling);

  // Step 3 : Bring result back to host
  hipMemcpy(numbers, d_numbers, size, hipMemcpyDeviceToHost);

  // Step 4 : Free device memory
  hipFree(d_numbers);

}
